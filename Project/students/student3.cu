#include "student3.hpp"

/*
* You have here to compute the segmented image from the filtered one.
* Calculations have to be done on Device using Thrust.
*
* @param in: input (filtered) image
* @param out: output (segmented) image
* @param threshold: thresholding value (remove the edges greater than it)
*/
float student3(const PPMBitmap& in, PPMBitmap& out, const int threshold) {
    return 0.f;
}