/*
* LW 5 - Atomics
* --------------------------
* Histogram equalization
*
* File: main.cu
*/

#include "app.hpp"

int main( int argc, char **argv ) {
	App *app = App::createInstance();
	app->launch( argc, argv );
	return ( EXIT_SUCCESS );
}
